#include "hip/hip_runtime.h"
/*********************************************************************/
//
// 02/01/2022: Revised Version for 32M bit adder with 32 bit blocks
//
/*********************************************************************/

#include "main.h"

//Touch these defines
#define input_size 8388608 // hex digits
#define block_size 32
#define verbose 0

//Do not touch these defines
#define digits (input_size+1)
#define bits (digits*4)
#define ngroups bits/block_size
#define nsections ngroups/block_size
#define nsupersections nsections/block_size
#define nsupersupersections nsupersections/block_size

//Global definitions of the various arrays used in steps for easy access
int gi[bits] = {0};
int pi[bits] = {0};
int ci[bits] = {0};

int sumi[bits] = {0};

int sumrca[bits] = {0};

//Integer array of inputs in binary form
int* bin1=NULL;
int* bin2=NULL;

//Character array of inputs in hex form
char* hex1=NULL;
char* hex2=NULL;


void read_input()
{
  char* in1 = (char *)calloc(input_size+1, sizeof(char));
  char* in2 = (char *)calloc(input_size+1, sizeof(char));

  if( 1 != scanf("%s", in1))
    {
      printf("Failed to read input 1\n");
      exit(-1);
    }
  if( 1 != scanf("%s", in2))
    {
      printf("Failed to read input 2\n");
      exit(-1);
    }

  hex1 = grab_slice_char(in1,0,input_size+1);
  hex2 = grab_slice_char(in2,0,input_size+1);

  free(in1);
  free(in2);
}

// Unrolled loop, each thread gets 1 bit.
__global__
void compute_gp_c(int* gi_c, int* pi_c, int* bin1_c, int* bin2_c){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < bits){
        gi_c[index] = bin1_c[index] & bin2_c[index];
        pi_c[index] = bin1_c[index] | bin2_c[index];
    }
}

// Unrolled loop, each thread gets 1 bit of ngroups, which reads a block_size
// chunk from gi and pi.
__global__
void compute_group_gp_c(int* ggj_c, int* gpj_c, int* gi_c, int* pi_c){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < ngroups){
        int jstart = index * block_size;
        int sum = 0;
        for(int i = 0; i < block_size; i++){
            int mult = gi_c[jstart + i];
            for(int ii = block_size-1; ii > i; ii--){
                mult &= pi_c[jstart + ii];
            }
            sum |= mult;
        }
        ggj_c[index] = sum;

        int mult = pi_c[jstart];
        for(int i = 1; i < block_size; i++){
            mult &= pi_c[jstart + i];
        }
        gpj_c[index] = mult;
    }

}

// Unrolled loop, each thread gets 1 bit of nsections, which reads a block_size
// chunk from ggj and gpj
__global__
void compute_section_gp_c(int* sgk_c, int* spk_c, int* ggj_c, int* gpj_c){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < nsections){
        int kstart = index * block_size;
        int sum = 0;
        for(int i = 0; i < block_size; i++){
            int mult = ggj_c[kstart + i];
            for(int ii = block_size-1; ii > i; ii--){
                mult &= gpj_c[kstart + ii];
            }
            sum |= mult;
        }
        sgk_c[index] = sum;

        int mult = gpj_c[kstart];
        for(int i = 1; i < block_size; i++){
            mult &= gpj_c[kstart + 1];
        }
        spk_c[index] = mult;
    }
}

// Unrolled loop, each thread gets 1 bit of nsupersections, which reads a block_size
// chunk from sgk and spk
__global__
void compute_super_section_gp_c(int* ssgl_c, int* sspl_c, int* sgk_c, int* spk_c){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < nsupersections){
        int lstart = index * block_size;
        int sum = 0;
        for(int i = 0; i < block_size; i++){
            int mult = sgk_c[lstart + i];
            for(int ii = block_size-1; ii > i; ii--){
                mult &= spk_c[lstart + ii];
            }
            sum |= mult;
        }
        ssgl_c[index] = sum;

        int mult = spk_c[lstart];
        for(int i = 1; i < block_size; i++){
            mult &= spk_c[lstart + 1];
        }
        sspl_c[index] = mult;
    }
}

// Unrolled loop, each thread gets 1 bit of nsupersupersections, which reads a block_size
// chunk from ssgl and sspl
__global__
void compute_super_super_section_gp_c(int* sssgm_c, int* ssspm_c, int* ssgl_c, int* sspl_c){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < nsupersupersections){
        int mstart = index * block_size;
        int sum = 0;
        for(int i = 0; i < block_size; i++){
            int mult = ssgl_c[mstart + i];
            for(int ii = block_size-1; ii > i; ii--){
                mult &= sspl_c[mstart + ii];
            }
            sum |= mult;
        }
        sssgm_c[index] = sum;

        int mult = sspl_c[mstart];
        for(int i = 1; i < block_size; i++){
            mult &= sspl_c[mstart + 1];
        }
        ssspm_c[index] = mult;
    }
}

// I dont think this one can be parelellized.
void compute_super_super_section_carry_c(int* ssscm_c, int* sssgm_c, int* ssspm_c){
    for(int m = 0; m < nsupersupersections; m++){
        int ssscmlast = 0;
        if(m == 0){
            ssscmlast = 0;
        } else{
            ssscmlast = ssscm_c[m-1];
        }
        ssscm_c[m] = sssgm_c[m] | (ssspm_c[m] & ssscmlast);
    }

}

// Each thread reads from ssscm and then iterates through 1 block
__global__
void compute_super_section_carry_c(int* sscl_c, int* ssgl_c, int* sspl_c, int* ssscm_c){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index == 0){
        sscl_c[0] = ssgl_c[0] | 0;
    }
    if(threadIdx.x < nsupersupersections){
        if(index != 0)
            sscl_c[index * block_size] = ssgl_c[index * block_size] | (sspl_c[index * block_size] & ssscm_c[index]);
        __syncthreads();
        index *= block_size;
        for(int l = 1; l < block_size; l++){
            if(index + l < nsupersections){
                sscl_c[index + l] = ssgl_c[index + l] | (sspl_c[index + l] & sscl_c[index + l -1]);
            }
        }
    }
}

// Each thread reads from sscl and iterates through 1 block
__global__
void compute_section_carry_c(int* sck_c, int* sgk_c, int* spk_c, int* sscl_c){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index == 0){
        sck_c[0] = sgk_c[0] | 0;
    }
    if(threadIdx.x < nsupersections){
        if(index != 0)
            sck_c[index * block_size] = sgk_c[index * block_size] | (spk_c[index * block_size] & sscl_c[index]);
        __syncthreads();
        index *= block_size;
        for(int k = 1; k < block_size; k++){
            if(index + k < nsections){
                sck_c[index + k] = sgk_c[index + k] | (spk_c[index + k] & sck_c[index + k - 1]);
            }
        }
    }
}

// Each thread reads from sck and iterates through 1 block
__global__
void compute_group_carry_c(int* gcj_c, int* ggj_c, int* gpj_c, int* sck_c){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index == 0){
        gcj_c[0] = ggc_c[0] | 0;
    }
    if(threadIdx.x < nsections){
        if(index != 0)
            gcj_c[index * block_size] = ggj_c[index * block_size] | (gpj_c[index * block_size] & sck_c[index]);
        __syncthreads();
        index *= block_size;
        for(int j = 1; j < block_size; j++){
            if(index + j < ngroups){
                gcj_c[index + j] = ggj_c[index + j] | (gpj_c[index + j] & gcj_c[index+j -1]);
            }
        }
    }
}

// Each thread reads from gcj and iterates through 1 block
__global__
void compute_carry_c(int* ci_c, int* gi_c, int* pi_c, int* gcj_c){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index == 0){
        ci_c[0] = gi_c[0] | 0;
    }
    if(threadIdx.x < ngroups){
        if(index != 0)
            ci_c[index * block_size] = gi_c[index * block_size] | (pi_c[index * block_size] & gcj_c[index-1]);
        __syncthreads();
        index *= block_size;
        for(int i = 1; i < block_size; i++){
            if(index + i < bits){
                ci_c[index + i] = gi_c[index + i] | (pi_c[index + i] & ci_c[index+i -1]);
            }
        }
    }
}

// Unrolled loop so each thread gets 1 bit to work with.
__global__
void compute_sum_c(int* sumi_c, int* bin1_c, int* bin2_c, int* ci_c){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index != 0){
        if(index < bits)
            sumi_c[index] = bin1_c[index] ^ bin2_c[index] ^ ci_c[index - 1];
    }
}

void cla()
{

    // Malloc bin1 and bin2 so GPU can read, and copy data over.
    int* bin1_cuda, *bin2_cuda;
    hipMallocManaged(&bin1_cuda, bits*sizeof(int));
    hipMallocManaged(&bin2_cuda, bits*sizeof(int));

    for(int i = 0; i < bits; i++){
        bin1_cuda[i] = bin1[i];
        bin2_cuda[i] = bin2[i];
    }

    int blockSize = 256;

    // Call all kernels, allocating arrays as needed

    int* gi_cuda, *pi_cuda;
    hipMallocManaged(&gi_cuda, bits*sizeof(int));
    hipMallocManaged(&pi_cuda, bits*sizeof(int));

    compute_gp_c<<<(bits + blockSize -1)/blockSize, blockSize>>>(gi_cuda, pi_cuda, bin1_cuda, bin2_cuda);

    int* ggj_cuda, *gpj_cuda;
    hipMallocManaged(&ggj_cuda, ngroups*sizeof(int));
    hipMallocManaged(&gpj_cuda, ngroups*sizeof(int));

    compute_group_gp_c<<<(ngroups + blockSize -1)/blockSize, blockSize>>>(ggj_cuda, gpj_cuda, gi_cuda, pi_cuda);

    int* sgk_cuda, *spk_cuda;
    hipMallocManaged(&sgk_cuda, nsections*sizeof(int));
    hipMallocManaged(&spk_cuda, nsections*sizeof(int));

    compute_section_gp_c<<<(nsections + blockSize -1)/blockSize, blockSize>>>(sgk_cuda, spk_cuda, ggj_cuda, gpj_cuda);

    int* ssgl_cuda, *sspl_cuda;
    hipMallocManaged(&ssgl_cuda, nsupersections*sizeof(int));
    hipMallocManaged(&sspl_cuda, nsupersections*sizeof(int));

    compute_super_section_gp_c<<<(nsupersections + blockSize -1)/blockSize, blockSize>>>(ssgl_cuda, sspl_cuda, sgk_cuda, spk_cuda);

    int* sssgm_cuda, *ssspm_cuda;
    hipMallocManaged(&sssgm_cuda, nsupersupersections*sizeof(int));
    hipMallocManaged(&ssspm_cuda, nsupersupersections*sizeof(int));

    compute_super_super_section_gp_c<<<(nsupersupersections + blockSize -1)/blockSize, blockSize>>>(sssgm_cuda, ssspm_cuda, ssgl_cuda, sspl_cuda);

    int* ssscm_cuda;
    hipMallocManaged(&ssscm_cuda, nsupersupersections*sizeof(int));
    compute_super_super_section_carry_c(ssscm_cuda, sssgm_cuda, ssspm_cuda);

    int* sscl_cuda;
    hipMallocManaged(&sscl_cuda, nsupersections*sizeof(int));
    compute_super_section_carry_c<<<(nsupersupersections + blockSize -1)/blockSize, blockSize>>>(sscl_cuda, ssgl_cuda, sspl_cuda, ssscm_cuda);

    int* sck_cuda;
    hipMallocManaged(&sck_cuda, nsections*sizeof(int));
    compute_section_carry_c<<<(nsupersections + blockSize -1)/blockSize, blockSize>>>(sck_cuda, sgk_cuda, spk_cuda, sscl_cuda);

    int* gcj_cuda;
    hipMallocManaged(&gcj_cuda, ngroups*sizeof(int));
    compute_group_carry_c<<<(nsections + blockSize -1)/blockSize, blockSize>>>(gcj_cuda, ggj_cuda, gpj_cuda, sck_cuda);

    int* ci_cuda;
    hipMallocManaged(&ci_cuda, bits*sizeof(int));
    compute_carry_c<<<(ngroups + blockSize -1)/blockSize, blockSize>>>(ci_cuda, gi_cuda, pi_cuda, gcj_cuda);

    int* sumi_cuda;
    hipMallocManaged(&sumi_cuda, bits*sizeof(int));
    compute_sum_c<<<(bits + blockSize -1)/blockSize, blockSize>>>(sumi_cuda, bin1_cuda, bin2_cuda, ci_cuda);


    // Synchronize and copy result into sumi from sumi_c
    hipDeviceSynchronize();

    sumi_cuda[0] = bin1_cuda[0] ^ bin2_cuda[0] ^ 0;

    for(int i = 0; i < bits; i++){
        sumi[i] = sumi_cuda[i];
    }

    // Free all cuda malloc'd memory.
    hipFree(bin1_cuda);
    hipFree(bin2_cuda);
    hipFree(gi_cuda);
    hipFree(pi_cuda);
    hipFree(ggj_cuda);
    hipFree(gpj_cuda);
    hipFree(sgk_cuda);
    hipFree(spk_cuda);
    hipFree(ssgl_cuda);
    hipFree(sspl_cuda);
    hipFree(sssgm_cuda);
    hipFree(ssspm_cuda);
    hipFree(ssscm_cuda);
    hipFree(sscl_cuda);
    hipFree(sck_cuda);
    hipFree(gcj_cuda);
    hipFree(ci_cuda);
    hipFree(sumi_cuda);
}

void ripple_carry_adder()
{
  int clast=0, cnext=0;

  for(int i = 0; i < bits; i++)
    {
      cnext = (bin1[i] & bin2[i]) | ((bin1[i] | bin2[i]) & clast);
      sumrca[i] = bin1[i] ^ bin2[i] ^ clast;
      clast = cnext;
    }
}

void check_cla_rca()
{
  for(int i = 0; i < bits; i++)
    {
      if( sumrca[i] != sumi[i] )
	{
	  printf("Check: Found sumrca[%d] = %d, not equal to sumi[%d] = %d - stopping check here!\n",
		 i, sumrca[i], i, sumi[i]);
	  printf("bin1[%d] = %d, bin2[%d]=%d, gi[%d]=%d, pi[%d]=%d, ci[%d]=%d, ci[%d]=%d\n",
		 i, bin1[i], i, bin2[i], i, gi[i], i, pi[i], i, ci[i], i-1, ci[i-1]);
	  return;
	}
    }
  printf("Check Complete: CLA and RCA are equal\n");
}

int main(int argc, char *argv[])
{
  int randomGenerateFlag = 1;
  int deterministic_seed = (1<<30) - 1;
  char* hexa=NULL;
  char* hexb=NULL;
  char* hexSum=NULL;
  char* int2str_result=NULL;
  unsigned long long start_time=clock_now(); // dummy clock reads to init
  unsigned long long end_time=clock_now();   // dummy clock reads to init

  if( nsupersupersections != block_size )
    {
      printf("Misconfigured CLA - nsupersupersections (%d) not equal to block_size (%d) \n",
	     nsupersupersections, block_size );
      return(-1);
    }

  if (argc == 2) {
    if (strcmp(argv[1], "-r") == 0)
      randomGenerateFlag = 1;
  }

  if (randomGenerateFlag == 0)
    {
      read_input();
    }
  else
    {
      srand( deterministic_seed );
      hex1 = generate_random_hex(input_size);
      hex2 = generate_random_hex(input_size);
    }

  hexa = prepend_non_sig_zero(hex1);
  hexb = prepend_non_sig_zero(hex2);
  hexa[digits] = '\0'; //double checking
  hexb[digits] = '\0';

  bin1 = gen_formated_binary_from_hex(hexa);
  bin2 = gen_formated_binary_from_hex(hexb);

  start_time = clock_now();
  cla();
  end_time = clock_now();

  printf("CLA Completed in %llu cycles\n", (end_time - start_time));

  start_time = clock_now();
  ripple_carry_adder();
  end_time = clock_now();

  printf("RCA Completed in %llu cycles\n", (end_time - start_time));

  check_cla_rca();

  if( verbose==1 )
    {
      int2str_result = int_to_string(sumi,bits);
      hexSum = revbinary_to_hex( int2str_result,bits);
    }

  // free inputs fields allocated in read_input or gen random calls
  free(int2str_result);
  free(hex1);
  free(hex2);

  // free bin conversion of hex inputs
  free(bin1);
  free(bin2);

  if( verbose==1 )
    {
      printf("Hex Input\n");
      printf("a   ");
      print_chararrayln(hexa);
      printf("b   ");
      print_chararrayln(hexb);
    }

  if ( verbose==1 )
    {
      printf("Hex Return\n");
      printf("sum =  ");
    }

  // free memory from prepend call
  free(hexa);
  free(hexb);

  if( verbose==1 )
    printf("%s\n",hexSum);

  free(hexSum);


  return 1;
}
